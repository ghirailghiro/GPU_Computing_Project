#include "hip/hip_runtime.h"
%%writefile gradient_computation.cu
#include <opencv2/opencv.hpp>
#include <fstream>
#include <sstream>
#include <iostream>
#include <vector>
#include <filesystem>
#include <chrono>
#include <tuple>

namespace fs = std::filesystem;

// Global variables
int cellSize_g;
int blockSize_g;
int descriptorSizeDimension_g;
int numBins_g;
int dimofimage_g;

std::vector<std::tuple<std::string, std::string, double, double, double>> memoryUsageLog;

std::string current_filename_g;

void logMemoryUsage(const std::string& label) {
    size_t free_memory, total_memory;
    hipMemGetInfo(&free_memory, &total_memory);
    size_t used_memory = total_memory - free_memory;

    double usedMemoryMB = used_memory / (1024.0 * 1024.0);
    double freeMemoryMB = free_memory / (1024.0 * 1024.0);
    double totalMemoryMB = total_memory / (1024.0 * 1024.0);

    memoryUsageLog.push_back(std::make_tuple(current_filename_g, label, usedMemoryMB, freeMemoryMB, totalMemoryMB));
}


void saveMemoryUsageLogToCSV(const std::string& filename) {
    std::ofstream file(filename);

    if (!file.is_open()) {
        std::cerr << "Error: Unable to open file " << filename << " for writing." << std::endl;
        return;
    }

    // Write CSV header
    file << "filename,Label,Used Memory (MB),Free Memory (MB),Total Memory (MB)\n";

    // Write each entry from the memoryUsageLog
    for (const auto& entry : memoryUsageLog) {
        file << std::get<0>(entry) << ","  // filename
             << std::get<1>(entry) << ","  // Label
             << std::get<2>(entry) << ","  // Used Memory (MB)
             << std::get<3>(entry) << ","  // Free Memory (MB)
             << std::get<4>(entry) << "\n";  // Total Memory (MB)
    }

    file.close();
}


void saveDescriptorAsCSVHeader(const std::vector<int>& descriptor, const std::string& filename, const std::string& label) {
    std::ofstream file(filename, std::ios::app);
    if (!file.is_open()) {
        std::cerr << "Error: Unable to open file " << filename << " for writing." << std::endl;
        return;
    }

    // Write the descriptor to the CSV file
    for (size_t i = 0; i < descriptor.size(); ++i) {
        file << "x" <<descriptor[i];
        if (i < descriptor.size() - 1) {
            file << ",";
        }
    }
    file << "," << "filename" << "," << "label" <<","<< "ExecTime" <<","<< "DimOfImage" << ","<<"ExecTimeMemLoading"<<"\n";
    file.close();
}


void saveDescriptorAsCSV(const std::vector<double>& descriptor, const std::string& filename,const std::string& path, int label,  double executionTime,  double executionTimeMemory) {
    std::ofstream file(filename, std::ios::app);
    if (!file.is_open()) {
        std::cerr << "Error: Unable to open file " << filename << " for writing." << std::endl;
        return;
    }

    // Write the descriptor to the CSV file
    for (size_t i = 0; i < descriptor.size(); ++i) {
        file << descriptor[i];
        if (i < descriptor.size() - 1) {
            file << ",";
        }
    }
    file << "," << path << "," << label << "," << executionTime <<","<< dimofimage_g <<","<< executionTimeMemory << "\n";
    file.close();
}

void computeGradients_seq(const cv::Mat& image, std::vector<float>& histograms, int cellSize, int numBins) {

    // Assuming image dimensions are reasonable for a grid of threads
    int width = image.cols;
    int height = image.rows;
    int numCellsX = width / cellSize;
    int numCellsY = height / cellSize;

    for (int idy = 0; idy < height; ++idy) {
        for (int idx = 0; idx < width; ++idx) {
            float G_x = 0, G_y = 0;
            if (idx > 0 && idx < width - 1) {
                G_x = static_cast<float>(image.at<uchar>(idy, idx + 1)) - static_cast<float>(image.at<uchar>(idy, idx - 1));
            }
            if (idy > 0 && idy < height - 1) {
                G_y = static_cast<float>(image.at<uchar>(idy + 1, idx)) - static_cast<float>(image.at<uchar>(idy - 1, idx));
            }

            float mag = std::sqrt(G_x * G_x + G_y * G_y);
            float orient = std::atan2(G_y, G_x);

            // Compute histogram bin for the current gradient
            int cellX = idx / cellSize;
            int cellY = idy / cellSize;
            int histIndex = cellY * numCellsX + cellX;

            // Assuming numBins represents the number of bins for the [0, 180] degree range
            float binWidth = M_PI / numBins;  // Bin width for [0, π] range

            // Calculate the gradient orientation as an unsigned angle
            if (orient < 0) {
                orient += M_PI;  // Normalize to [0, π] range, example if we have -45°, we add 180° to get 135°
            }
            // Calculate the bin index
            int bin = static_cast<int>(std::round(orient / binWidth)) % numBins;

            int final_index = histIndex * numBins + bin;
            if (final_index >= histograms.size()) {
                std::cout << "Index out of bounds : " << final_index << std::endl;
            }else{
                histograms[final_index] += mag;
            }

        }
    }

    std::cout << "Ending computeGradients seq" << std::endl;
}

__global__ void computeGradients(unsigned char* image, float *d_histograms, int width, int height, int cellSize,float binWidth, int numBins, int histSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int indexCurrent = idy * width + idx;

    if (idx >= width || idy >= height) return; // Boundary check

    float G_x = 0;
    // Compute gradients in x and y directions.
    //The conditional statements check if the current pixel is within the image boundaries.
    //If the pixel is not on the left or right edge of the image, the gradient in the x-direction is computed by subtracting the pixel value on the left from the pixel value on the right.
    if (idx > 0 && idx < width - 1) {
        G_x = (float)image[indexCurrent + 1] - (float)image[indexCurrent - 1];
    }

    float G_y = 0;
    if (idy > 0 && idy < height - 1) {
        G_y = (float)image[(idy + 1) * width + idx] - (float)image[(idy - 1) * width + idx]; // To Do: Capire formula??
    }

    float d_magnitude_var = sqrtf(G_x * G_x + G_y * G_y);
    float d_orientation_var = atan2f(G_y, G_x);

    // Compute histogram bin for the current gradient
    int cellX = idx / cellSize;
    int cellY = idy / cellSize;
    //The division (width / cellSize) calculates the ratio between the width of the grid and the size of each cell.
    //This ratio determines the number of cells that can fit horizontally in the grid.
    //By multiplying this ratio with cellY, we obtain the number of cells that can fit vertically up to the Y-coordinate cellY.
    //Finally, the expression cellY * (width / cellSize) + cellX adds the X-coordinate cellX to the previously calculated value.
    //This addition determines the absolute position of a cell within the grid, considering both its X and Y coordinates.
    int histIndex = cellY * (width / cellSize) + cellX;

    //The following formula calculates the bin index for the current orientation value:
    /*1. `d_orientation[indexCurrent]`: This is a variable or an array element that holds the orientation value at the `indexCurrent` position.
        The orientation value is likely in radians.

    2. `M_PI`: This is a constant defined in the C++ math library that represents the value of pi (π).
        It is used to shift the orientation value by π radians.

    3. `(d_orientation[indexCurrent] + M_PI)`: This expression adds the orientation value to π, effectively shifting the range of values from [-π, π] to [0, 2π].

    4. `binWidth`: This is likely another variable or constant that represents the width of each bin. Bins are used to categorize or group values within a certain range.

    5. `(d_orientation[indexCurrent] + M_PI) / binWidth`: This expression divides the shifted orientation value by the bin width. The result is a floating-point number that represents the bin index.

    6. `floor((d_orientation[indexCurrent] + M_PI) / binWidth)`: The `floor()` function is used to round down the floating-point bin index to the nearest integer. This ensures that the bin index is an integer value.

    */
    // Calculate the gradient orientation as an unsigned angle
    if (d_orientation_var < 0) {
        d_orientation_var += M_PI;  // Normalize to [0, π] range, example if we have -45°, we add 180° to get 135°
    }
    // Calculate the bin index
    int bin = __float2int_rn(d_orientation_var / binWidth) % numBins;

    int final_index = histIndex * numBins + bin;
    if (final_index >= histSize) {
        printf("Index out of bounds: %d\n", final_index);
    }else{
      atomicAdd(&d_histograms[final_index], d_magnitude_var);
    }
}

std::vector<double> computeDescriptorsCUDA(const cv::Mat& image, double& executionTime, double& LoadingInMemoryTime) {
    int numCellsX = image.cols / cellSize_g;
    int numCellsY = image.rows / cellSize_g;
    hipDeviceSynchronize();
    logMemoryUsage("Before memory allocation");
    auto startMemory = std::chrono::high_resolution_clock::now();
    unsigned char* d_image;
    size_t imageSize = image.total() * image.elemSize();
    hipError_t status = hipMalloc(&d_image, imageSize);
    // Allocate memory for orientation
    if (status != hipSuccess) {
        // Handle error (e.g., printing an error message and exiting)
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(status));
        exit(EXIT_FAILURE);
    }
    // Initialize d_orientation to zero
    status = hipMemcpy(d_image, image.data, imageSize, hipMemcpyHostToDevice);
    if (status != hipSuccess) {
        // Handle error
        fprintf(stderr, "hipMemset failed: %s\n", hipGetErrorString(status));
        exit(EXIT_FAILURE);
    }
    // hist size is the number of cells in the x and y direction times 9 bins per cell
    size_t histSize = numCellsX * numCellsY * numBins_g * sizeof(float);
    float* d_histograms; //device histograms
    // Allocate memory for histograms
    status = hipMalloc((void **)&d_histograms, histSize);
    if (status != hipSuccess) {
        // Handle error
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(status));
        exit(EXIT_FAILURE);
    }
    // Initialize histograms to zero
    status = hipMemset(d_histograms, 0, histSize);
    if (status != hipSuccess) {
        // Handle error
        fprintf(stderr, "hipMemset failed: %s\n", hipGetErrorString(status));
        exit(EXIT_FAILURE);
    }
    auto endMemory = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsedMemory = endMemory - startMemory;
    LoadingInMemoryTime = elapsedMemory.count();
    hipDeviceSynchronize();
    logMemoryUsage("After allocating memory");

    std::cout << "Loading Memory Time Step"<< LoadingInMemoryTime << std::endl;

    // Assuming image dimensions are reasonable for a blocksize 16x16
    //By dividing the image dimensions by the block size and rounding up to the nearest integer, the grid size is determined.
    //The -1 in the calculation is used to handle cases where the image dimensions are not evenly divisible by the block size.
    //This ensures that any remaining pixels are included in the grid.
    dim3 blockSize(16, 16);
    dim3 gridSize((image.cols + blockSize.x - 1) / blockSize.x,
                  (image.rows + blockSize.y - 1) / blockSize.y);

    // Bin width for [0, π] range
    float binWidth = M_PI / numBins_g;
    int histSize_vec = numCellsX * numCellsY * numBins_g;

    auto start = std::chrono::high_resolution_clock::now();
    // Launch the kernel
    computeGradients<<<gridSize, blockSize>>>(d_image, d_histograms, image.cols, image.rows, cellSize_g, binWidth, numBins_g, histSize_vec);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;
    executionTime = elapsed.count();

    // Transfer histogram data from device to host
    auto startMemoryFromDevice = std::chrono::high_resolution_clock::now();
    float* h_histograms = new float[numCellsX * numCellsY * numBins_g];
    status = hipMemcpy(h_histograms, d_histograms, histSize, hipMemcpyDeviceToHost);
    if (status != hipSuccess) {
        // Handle error
        fprintf(stderr, "hipMemcpy failed: %s\n", hipGetErrorString(status));
        exit(EXIT_FAILURE);
    }
    auto endMemoryFromDevice = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsedMemoryFromDevice = endMemoryFromDevice - startMemoryFromDevice;
    LoadingInMemoryTime += elapsedMemoryFromDevice.count();

    std::cout << "Loading Memory Time Final"<< LoadingInMemoryTime << std::endl;

    // Block Formation and Descriptor Computation with Block-Level Normalization
    std::vector<double> descriptor;
    for (int i = 0; i <= numCellsY - blockSize_g; ++i) {
        for (int j = 0; j <= numCellsX - blockSize_g; ++j) {
            // Step 1: Calculate the L2-norm for the block
            double blockNorm = 0;
            for (int y = i; y < i + blockSize_g; ++y) {
                for (int x = j; x < j + blockSize_g; ++x) {
                    for (int k = 0; k < numBins_g; ++k) {
                        float histValue = h_histograms[(y * numCellsX + x) * numBins_g + k];
                        blockNorm += histValue * histValue;
                    }
                }
            }
            blockNorm = sqrtf((blockNorm*blockNorm) + 1e-6 * 1e-6); // Small constant to avoid division by zero

            // Step 2: Normalize the histograms within the block
            for (int y = i; y < i + blockSize_g; ++y) {
                for (int x = j; x < j + blockSize_g; ++x) {
                    for (int k = 0; k < numBins_g; ++k) {
                        double normalizedValue = h_histograms[(y * numCellsX + x) * numBins_g + k] / blockNorm;
                        descriptor.push_back(normalizedValue);
                    }
                }
            }
        }
    }

    hipFree(d_image);
    delete[] h_histograms;
    hipFree(d_histograms);
    hipDeviceSynchronize();
    logMemoryUsage("After freeing all memory");


    return descriptor;
}

std::vector<double> computeDescriptorsSeq(const cv::Mat& image, double& executionTime) {

    int numCellsX = image.cols / cellSize_g;
    int numCellsY = image.rows / cellSize_g;

    // Allocate memory for histograms
    std::vector<float> histograms(numCellsX * numCellsY * numBins_g, 0.0f);
    auto start = std::chrono::high_resolution_clock::now();
    computeGradients_seq(image, histograms, cellSize_g, numBins_g);
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;
    executionTime = elapsed.count();

    std::cout << "Before computing descriptor seq" << std::endl;

    // Block Formation and Descriptor Computation with Block-Level Normalization
    std::vector<double> descriptor;
    for (int i = 0; i <= numCellsY - blockSize_g; ++i) {
        for (int j = 0; j <= numCellsX - blockSize_g; ++j) {
            // Step 1: Calculate the L2-norm for the block
            double blockNorm = 0;
            for (int y = i; y < i + blockSize_g; ++y) {
                for (int x = j; x < j + blockSize_g; ++x) {
                    for (int k = 0; k < numBins_g; ++k) {
                        float histValue = histograms[(y * numCellsX + x) * numBins_g + k];
                        blockNorm += histValue * histValue;
                    }
                }
            }

            blockNorm = sqrtf((blockNorm*blockNorm) + 1e-6 * 1e-6); // Small constant to avoid division by zero
            // Step 2: Normalize the histograms within the block
            for (int y = i; y < i + blockSize_g; ++y) {
                for (int x = j; x < j + blockSize_g; ++x) {
                    for (int k = 0; k < numBins_g; ++k) {
                        double normalizedValue = histograms[(y * numCellsX + x) * numBins_g + k] / blockNorm;
                        descriptor.push_back(normalizedValue);
                    }
                }
            }
        }
    }

    std::cout << "Ending computing descriptor seq" << std::endl;


    return descriptor;
}

std::vector<double> computeDescriptors(const std::string& image_path, double& executionTime, double& LoadingInMemoryTime, bool cudaAccelerated = true) {

    cv::Mat imageBeforeResize = cv::imread(image_path, cv::IMREAD_GRAYSCALE);
    cv::Mat image;
    cv::resize(imageBeforeResize, image, cv::Size(dimofimage_g, dimofimage_g));
    if(image.empty()) {
        std::cerr << "Failed to load image." << std::endl;
        return std::vector<double>();
    }
    std::vector<double> descriptor;
    if(cudaAccelerated) {
        descriptor = computeDescriptorsCUDA(image, executionTime, LoadingInMemoryTime);
    } else {
        descriptor = computeDescriptorsSeq(image, executionTime);
    }
    std::cout << "End COMPUTE DESCRIPTOR ----------" << std::endl;

    return descriptor;
}

int main(int argc, char** argv) {
     if (argc != 6) {
        std::cerr << "Usage: " << argv[0] << " <cellSize> <blockSize> <numBins> <outputFile> <dimofimage>" << std::endl;
        return 1;
    }

    cellSize_g = std::stoi(argv[1]);
    blockSize_g = std::stoi(argv[2]);
    numBins_g = std::stoi(argv[3]);
    std::string outputFile = argv[4];
    dimofimage_g = std::stoi(argv[5]);//224

    // Calcute numbers of cells in x and y direction
    int numCellsX = dimofimage_g / cellSize_g;
    int numCellsY = dimofimage_g / cellSize_g;
    /*This is how we calculate the descriptorSizeDimension:

    1. `(numCellsY - blockSize + 1)` calculates the number of blocks in the Y direction.
        Here, `numCellsY` represents the total number of cells in the Y direction, and `blockSize` represents the size of each block.
        By subtracting `blockSize - 1` from `numCellsY`, we account for the overlapping blocks.

    2. `(numCellsX - blockSize + 1)` calculates the number of blocks in the X direction.
        Similar to the previous step, `numCellsX` represents the total number of cells in the X direction, and `blockSize` represents the size of each block.
        Again, we subtract `blockSize - 1` to account for the overlapping blocks.

    3. `blockSize * blockSize` calculates the number of cells within each block.
        Since the blocks are square, we multiply the `blockSize` by itself to get the total number of cells in a block.

    4. `numBins` represents the number of bins used for the descriptor. Each cell in the histogram contains `numBins` values.
        By multiplying all these values together, we get the total size of the descriptor.
        The descriptor size is the product of the number of blocks in the X and Y directions, the number of cells within each block, and the number of bins.
    */
    descriptorSizeDimension_g = (numCellsY - blockSize_g + 1) * (numCellsX - blockSize_g + 1) * blockSize_g * blockSize_g * numBins_g;

    std::string folder_path = "/content/drive/My Drive/GPU Computing/human detection dataset/1"; // Change this to your folder path
    std::vector<int> header;
    for (int i=1; i <= descriptorSizeDimension_g; ++i){
      header.push_back(i);
    }
    std::cout << "Descriptor size : " << descriptorSizeDimension_g << std::endl;
    std::string seq_file = outputFile+"_seq.csv";
    std::string cuda_file = outputFile+"_cuda.csv";
    std::cout << seq_file << std::endl;
    std::cout << cuda_file << std::endl;
    saveDescriptorAsCSVHeader(header, seq_file, "label");
    saveDescriptorAsCSVHeader(header, cuda_file, "label");
    header.clear();
    //Iterate on images where a human is present
    for (const auto& entry : fs::directory_iterator(folder_path)) {
        std::string file_path = entry.path().string();
        std::cout << "Processing image: " << file_path << std::endl;

        double executionTimeCuda = 0.0;
        double executionTimeSeq = 0.0;
        double loadingTimeInMemoryCuda = 0.0;
        double loadingTimeInMemorySeq = 0.0;
        std::vector<double> descriptor = computeDescriptors(file_path, executionTimeCuda, loadingTimeInMemoryCuda);
        std::vector<double> descriptor_seq = computeDescriptors(file_path, executionTimeSeq, loadingTimeInMemorySeq, false);

        current_filename_g = file_path;
        if (descriptor_seq.empty()) {
            std::cout << "Vector is empty" << std::endl;
        } else {
            int label = 1;
            std::cout << descriptor_seq.size() << std::endl;
            std::cout << descriptor[0] << std::endl;
            std::cout << descriptor_seq[0] << std::endl;
            saveDescriptorAsCSV(descriptor, cuda_file, file_path, label, executionTimeCuda, loadingTimeInMemoryCuda);
            saveDescriptorAsCSV(descriptor_seq, seq_file, file_path, label, executionTimeSeq, loadingTimeInMemorySeq);
        }
        descriptor.clear();
        descriptor_seq.clear();
    }

      //Iterate on images where a human is NOT present
      folder_path = "/content/drive/My Drive/GPU Computing/human detection dataset/0";
      for (const auto& entry : fs::directory_iterator(folder_path)) {
        std::string file_path = entry.path().string();
        std::cout << "Processing image: " << file_path << std::endl;

        double executionTimeCuda = 0.0;
        double executionTimeSeq = 0.0;
        double loadingTimeInMemoryCuda = 0.0;
        double loadingTimeInMemorySeq = 0.0;
        std::vector<double> descriptor = computeDescriptors(file_path, executionTimeCuda, loadingTimeInMemoryCuda);
        std::vector<double> descriptor_seq = computeDescriptors(file_path, executionTimeSeq, loadingTimeInMemorySeq, false);        //std::vector<double> descriptor_seq;
        current_filename_g = file_path;
        if (descriptor_seq.empty()) {
            std::cout << "Vector is empty" << std::endl;
        } else {
            int label = 0;
            std::cout << descriptor_seq.size() << std::endl;
            saveDescriptorAsCSV(descriptor, cuda_file, file_path, label, executionTimeCuda, loadingTimeInMemoryCuda);
            saveDescriptorAsCSV(descriptor_seq, seq_file, file_path, label, executionTimeSeq, loadingTimeInMemorySeq);
        }
        descriptor.clear();
        descriptor_seq.clear();
    }

    saveMemoryUsageLogToCSV(outputFile+"_memory_usage_cuda_log.csv");

    return 0;
}